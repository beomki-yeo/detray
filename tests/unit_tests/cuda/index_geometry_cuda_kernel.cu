#include "hip/hip_runtime.h"
/** Detray library, part of the ACTS project (R&D line)
 *
 * (c) 2021 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

#include <vecmem/containers/device_vector.hpp>

#include "detray/definitions/cuda_defs.hpp"
#include "index_geometry_cuda_kernel.hpp"

namespace detray {

__global__ void index_geometry_test_kernel(
    index_geometry_data<> geometry_data,
    vecmem::data::vector_view<typename geometry::volume_type> output_data) {

    index_geometry<vecmem::device_vector> g(geometry_data);

    vecmem::device_vector<typename geometry::volume_type> output_device(
        output_data);

    for (unsigned int i = 0; i < g.n_volumes(); i++) {
        output_device[i] = g.volume_by_index(i);
    }
}

void index_geometry_test(
    index_geometry_data<>& geometry_data,
    vecmem::data::vector_view<typename geometry::volume_type>& output_data) {

    int block_dim = 1;
    int thread_dim = 1;

    // run the kernel
    index_geometry_test_kernel<<<block_dim, thread_dim>>>(geometry_data,
                                                          output_data);

    // cuda error check
    DETRAY_CUDA_ERROR_CHECK(hipGetLastError());
    DETRAY_CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

}  // namespace detray
